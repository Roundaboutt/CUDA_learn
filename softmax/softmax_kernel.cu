#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

//---------------------------------------------------------------------------------------------------------------

// CPU中实现
void softmax_cpu(float* output, float* input, int N, int C){
    float maxval = -INFINITY;
    for (int i = 0;i < N; i++){
        const float* inp_row = input + C * i;

        for (int j = 0;j < C; j++){
            maxval = fmaxf(maxval, inp_row[j]);
        }
    }

    for (int i = 0;i < N; i++){
        const float* inp_row = input + C * i;
        float* out_row = output + C * i;
        
        float sum = 0.0f;
        for (int j = 0;j < C; j++){
            out_row[j] = expf(inp_row[j] - maxval);
            sum += out_row[j];
        }


        float norm = 1.0f / sum;
        for (int j = 0;j < C; j++){
            out_row[j] *= norm;
        }
    }


}

//---------------------------------------------------------------------------------------------------------------

// N个向量,每个向量中有C个元素
__global__ void softmax_kernel1(float* output, float* input, int N, int C){

    // N个线程块,每个线程块负责一个向量的计算
    // 每个线程块中只有一个线程计算
    // id:全局线程索引
    int id = blockIdx.x * blockDim.x + threadIdx.x; // id:[0, N-1)  blockIdx:[0, N-1)  threadIdx:0
    if (id < N){
        const float* input_row = input + id*C;
        float* output_row = output + id*C;

        float maxval = -INFINITY;
        for (int i = 0;i < C;i++){
            if (input_row[i] > maxval){
                maxval = input_row[i];
            }
        }

        float sum = 0.f;
        for (int j = 0;j < C;j++){
            output_row[j] = expf(input_row[j] - maxval);
            sum += output_row[j];
        }

        for (int j = 0;j < C;j++){
            output_row[j] /= sum;
        }
    }

}
//---------------------------------------------------------------------------------------------------------------

//利用规约操作和共享显存优化
__global__ void softmax_kernel2(float* output, float* input, int N, int C){

    // 声明共享显存
    extern __shared__ float shared[];
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int block_size = blockDim.x;
    const float* input_row = input + bid * C;
    
    float maxval = -INFINITY;

    // 每个线程处理以下下标的数据:tid, tid + block_size, tid + 2 * block_size ...
    for (int i = tid; i < C;i += block_size){
        // 这里为什么不需要同步? 因为线程之间没有相互依赖
        maxval = fmaxf(maxval, input_row[i]);
    }
    // 每个线程负责的所有元素中的最大值 写入共享显存
    shared[tid] = maxval;
    __syncthreads();

    // 从局部最大值中找出全局最大值
    for (int stride = block_size / 2; stride >= 1; stride /= 2){

        // 等上一轮所有线程都完成之后再比较
        __syncthreads();
        if (tid < stride){
            shared[tid] = fmaxf(shared[tid], shared[tid + stride]);
        }
    }

    __syncthreads();
    float offset = shared[0];   // 最终的全局最大值

    for (int i = tid;i < C;i += block_size){
        output[bid * C + i] = expf(input_row[i] - offset);
    }

    const float* output_row = output + bid * C;
    float sumval = 0.0f;
    for (int i = tid;i < C;i += block_size){
        sumval += output_row[i];
    }

    // 索引为tid的线程所有负责元素的和
    shared[tid] = sumval;
    __syncthreads();

    // 规约计算全局和
    for (int stride = block_size / 2; stride >= 1; stride /= 2){
        __syncthreads();
        if (tid < stride){
            shared[tid] += shared[tid + stride];            
        }
    }

    __syncthreads();
    float sum = shared[0];
    for (int i = tid;i < C;i += block_size){
        output[bid * C + i] = output_row[i] / sum;
    }
}
//---------------------------------------------------------------------------------------------------------------

//利用warp洗牌指令优化
__device__ float warpReduceSum(float val){
    for (int offset = 16; offset >= 1; offset /= 2){
        val += __shfl_down_sync(0xffffffff, val, offset, 32);
    }
    return val;
}

__device__ float warpReduceMax(float val){
    for (int offset = 16; offset >= 1; offset /= 2){
        val = fmaxf(__shfl_down_sync(0xffffffff, val, offset, 32), val);
    }
    return val;
}


__global__ void softmax_kernel3(float* output, float* input, int N, int C){
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int block_size = blockDim.x;
    
    const float* input_row = input + bid * C;
    float maxval = -INFINITY;
    for (int i = tid; i < C; i += block_size){
        maxval = fmaxf(maxval, input_row[i]);
    }

    maxval = warpReduceMax(maxval);
    float offset = __shfl_sync(0xffffffff, maxval, 0);

    float* output_row = output + bid * C;
    for (int i = tid;i < C; i += block_size){
        output_row[i] = expf(input_row[i] - offset);
    }

    float sumval = 0.0f;
    for (int i = tid; i < C; i += block_size){
        sumval += output_row[i];
    }
    sumval = warpReduceSum(sumval);

    float sum = __shfl_sync(0xffffffff, sumval, 0);
    for (int i = tid;i < C; i += block_size){
        output_row[i] /= sum;
    }

}





//---------------------------------------------------------------------------------------------------------------
int main(){
    int N = 32;
    int C = 4096;
    size_t elemCount = N * C;

    float* input = (float*)malloc(sizeof(float)*elemCount);
    float* output = (float*)malloc(sizeof(float)*elemCount);

    for (int n = 0;n < N;n++){
        for(int c = 0;c < C;c++){
            input[n*C + c] = float(0);
        }
    }

    softmax_cpu(output, input, N, C);


    for (int i = 0;i < 16; i++){
        printf("%.10f\n", output[i]);
    }
    printf("--------------------------------------------------\n");
    float* d_input,* d_output;
    hipMalloc((void**)&d_input, elemCount*sizeof(float));
    hipMalloc((void**)&d_output, elemCount*sizeof(float));
    hipMemcpy(d_input, input, elemCount*sizeof(float), hipMemcpyHostToDevice);


    int blockSize = 32; 
    int numBlocks = N;  // N个线程块,每个线程块负责一个向量

    softmax_kernel3<<<numBlocks, blockSize>>>(d_output, d_input, N, C);

    hipMemcpy(output, d_output, elemCount*sizeof(float), hipMemcpyDeviceToHost);


    for (int i = 0;i < 16; i++){
        printf("%.10f\n", output[i]);
    }

}