#include<iostream>
#include<hip/hip_runtime.h>
#include<random>

#define FULLMASK 0xFFFFFFFF

__device__ float BlockReduce(float val){
    const int tid = threadIdx.x;
    const int warpID = tid / warpSize;
    const int laneID = tid % warpSize;
    
    #pragma unroll
    for (int offset = warpSize / 2; offset > 0; offset >>= 1){
        val += __shfl_down_sync(FULLMASK, val, offset);
    }

    __shared__ float warpShared[32];
    if (laneID == 0)
        warpShared[warpID] = val;
    
    __syncthreads();

    if (warpID == 0){
        val = warpShared[laneID];
        #pragma unroll
        for (int offset = warpSize / 2; offset > 0; offset >>= 1){
            val += __shfl_down_sync(FULLMASK, val, offset);
        }

    }

    return val;
}

__global__ void rmsnorm_v2(float* input, float* output, int batch, int size, float* weight, float eps){
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;

    if  (bid >= batch) return;
    
    float* input_start = input + size * bid;
    float* output_start = output + size * bid;

    constexpr int pack_size = 4;
    const int pack_num = size / pack_size;
    // 剩余部分:(pack_off, size)
    const int pack_off = pack_num * pack_size;

    float sum = 0.f;
    float4* in_pack = reinterpret_cast<float4*>(input_start);
    for (int i = tid; i < pack_num; i += blockDim.x){
        float4 in_float4 = *(in_pack + i);
        sum += in_float4.x * in_float4.x;
        sum += in_float4.y * in_float4.y;
        sum += in_float4.z * in_float4.z;
        sum += in_float4.w * in_float4.w;
    }

    for (int i = pack_off + tid; i < size; i += blockDim.x){
        sum += input_start[i] * input_start[i];
    }

    sum = BlockReduce(sum);
    __shared__ float shared_val;
    if (tid == 0) shared_val = sum;

    __syncthreads();

    sum = shared_val;

    const float scale = rsqrtf(sum / static_cast<float>(size) + eps);
    float4* out_pack = reinterpret_cast<float4*>(output_start);
    float4* wei_pack = reinterpret_cast<float4*>(weight);
    for (int i = tid; i < pack_num; i += blockDim.x){
        float4 wei_float4 = *(wei_pack + i);
        float4 in_float4 = *(in_pack + i);

        *(out_pack + i)= make_float4(
            scale * wei_float4.x * in_float4.x,
            scale * wei_float4.y * in_float4.y,
            scale * wei_float4.z * in_float4.z,
            scale * wei_float4.w * in_float4.w
        );
    }

    for (int i = pack_off + tid; i < size; i += blockDim.x){
        output_start[i] = scale * weight[i] * input_start[i];
    }
}

bool isMatch(float* output_cpu, float* output_gpu, int elemCount){
    for (int i = 0; i < elemCount; i++){
        if (abs(output_cpu[i] - output_gpu[i]) > 1e-5){
            return false;
        }
    }
    return true;
}


void rmsnorm_cpu(float* input, float* output, int batch, int size, float* weight, float eps){
    for (int i = 0; i < batch; i++){
        float* input_start = input + size * i;
        float* output_start = output + size * i;

        float sum = 0.f;
        for (int j = 0; j < size; j++){
            sum += input_start[j] * input_start[j];
        }

        float rms = 1.f / std::sqrt(sum / static_cast<float>(size) + eps);

        for (int j = 0; j < size; j++){
            output_start[j] = input_start[j] * weight[j] * rms;
        }
    }
}


int main(){
    const int batch = 16;
    const int size = 1024;
    const float eps = 1e-6;
    const int elemCount = batch * size;
    const int numBytes = elemCount * sizeof(float);

    float* h_input,* h_output_cpu,* h_output_gpu,* h_weight;
    h_input = (float*)malloc(numBytes);
    h_output_cpu = (float*)malloc(numBytes);
    h_output_gpu = (float*)malloc(numBytes);
    h_weight = (float*)malloc(sizeof(float) * size);

    std::random_device rd;
    std::mt19937 gen(rd());
    std::normal_distribution<float> dis(0.0f, 1.0f);

    for (int i = 0; i < elemCount; i++){
        h_input[i] = dis(gen);
    }

    for (int i = 0; i < size; i++){
        h_weight[i] = dis(gen);
    }

    rmsnorm_cpu(h_input, h_output_cpu, batch, size, h_weight, eps);

    // ------------------------------

    float* d_input,* d_output,* d_weight;
    hipMalloc((void**)&d_input, numBytes);
    hipMalloc((void**)&d_output, numBytes);
    hipMalloc((void**)&d_weight, sizeof(float) * size);

    hipMemcpy(d_input, h_input, numBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_weight, h_weight, sizeof(float) * size, hipMemcpyHostToDevice);

    const int blocksize = 1024;
    const int gridsize = batch;
    dim3 grid(gridsize);
    dim3 block(blocksize);

    rmsnorm_v2<<<grid, block>>>(d_input, d_output, batch, size, d_weight, eps);

    hipMemcpy(h_output_gpu, d_output, numBytes, hipMemcpyDeviceToHost);

    if (isMatch(h_output_cpu, h_output_gpu, elemCount)){
        std::cout << "Match!" << std::endl;
    }
    else{
        std::cout << "Not Match!" << std::endl;
    }

}