#include<iostream>
#include<chrono>
#include"hip/hip_runtime.h"
#include<hipblas.h>
#include<vector>

template <const int BM, const int BN, const int BK, const int TM, const int TN>
__global__ void mysgemm_v3(int M, int N, int K, float alpha, float* A, float* B, float beta, float* C){
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // 一个block内的线程数
    // 每个线程负责处理一个 TM * TN 大小的数据tile
    int block_row_thread = BN / TN;
    int block_col_thread = BM / TM;
    int thread_num = block_row_thread * block_col_thread;

    // 当前线程在块内计算的起始地址
    int ty = (threadIdx.x / block_row_thread) * TM;
    int tx = (threadIdx.x % block_row_thread) * TN;

    __shared__ float As[BM * BK];
    __shared__ float Bs[BK * BN];

    A = &A[by * K * BM];
    B = &B[bx * BN];
    C = &C[BM * N * by + bx * BN];

    
    int a_tile_row = threadIdx.x / BK;      // 当前线程要搬到A tile的第几行
    int a_tile_col = threadIdx.x % BK;
    int a_tile_stride = thread_num / BK;    // 一个线程在for循环里负责不止一行, 要隔stride行再搬

    int b_tile_row = threadIdx.x / BN;      // 当前线程要搬到B tile的第几行
    int b_tile_col = threadIdx.x % BN;
    int b_tile_stride = thread_num / BN;

    float temp[TM][TN] = {0.};

#pragma unroll
    for (int k = 0; k < K; k += BK){

#pragma unroll
        for (int i = 0; i < BM; i += a_tile_stride){
            As[(a_tile_row + i) * BK + a_tile_col] = A[(a_tile_row + i) * K + a_tile_col];
        }

#pragma unroll
        for (int i = 0; i < BK; i += b_tile_stride){
            Bs[(b_tile_row + i) * BN + b_tile_col] = B[(b_tile_row + i) * N + b_tile_col];
        }        

        __syncthreads();
        A += BK;
        B += BK * N;

#pragma unroll
        for (int k1 = 0; k1 < BK; k1++){
#pragma unroll
            for (int i = 0; i < TM ; i++){
                for (int j = 0; j < TN; j++){
                    temp[i][j] += As[(ty + i) * BK + k1] * Bs[tx + j + BN * k1];
                }
            }
        }
        __syncthreads();
    }
#pragma unroll
    for (int i = 0; i < TM; i++){
        for (int j = 0; j < TN; j++){
            C[(ty + i) * N + tx + j] = alpha * temp[i][j] + beta * C[(ty + i) * N + tx + j];
        }
    }
    
}

int main(){
    std::vector<int> sizes = {128, 256, 512, 1024, 2048, 4096, 8192};
    for (int N:sizes){
        size_t elemCount = N * N;
        std::cout<<"------------------------Testing size: "<< N <<"------------------------"<< std::endl;

        size_t numBytes = elemCount * sizeof(float);
        float* A = (float*)malloc(numBytes);
        float* B = (float*)malloc(numBytes);
        float* C_cublas = (float*)malloc(numBytes);
        float* C_v1 = (float*)malloc(numBytes);

        float* d_A,* d_B, * d_C_v1;
        hipMalloc(&d_A, numBytes);
        hipMalloc(&d_B, numBytes);
        hipMalloc(&d_C_v1, numBytes);

        try{
            for (int i = 0; i < elemCount; i++){
                A[i] = 1.0f;
                B[i] = 2.0f;
            }

            hipMemcpy(d_A, A, numBytes, hipMemcpyHostToDevice);
            hipMemcpy(d_B, B, numBytes, hipMemcpyHostToDevice);
            
            hipblasHandle_t handle;  //定义句柄, 用于记录状态
            hipblasCreate(&handle);  // 创建cuBLAS上下文

            float alpha = 1.f;
            float beta = 0.f;
            

            /*------------------------cublas计算/*------------------------*/
            hipEvent_t start, end;
            hipEventCreate(&start);
            hipEventCreate(&end);

            int warmup_time = 10;
            for (int i = 0; i < warmup_time; i++){
                hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_B, N, d_A, N, &beta, d_C_v1, N);
            }

            hipDeviceSynchronize();

            int repeat_time = 5;
            hipEventRecord(start);
            for (int i = 0; i < repeat_time; i++){
                hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_B, N, d_A, N, &beta, d_C_v1, N);
            }
            hipEventRecord(end);
            hipEventSynchronize(end);

            float cublas_time = 0;
            hipEventElapsedTime(&cublas_time, start, end);
            
            hipMemcpy(C_cublas, d_C_v1, numBytes, hipMemcpyDeviceToHost);
            std::cout<<"cublas time:"<<cublas_time<<"ms"<<std::endl;

            hipDeviceSynchronize();

            /*------------------------v3计算------------------------*/
            dim3 threads(256); // 
            dim3 blocks((N + 128 - 1) / 128, (N + 128 - 1) / 128);
            
            for (int i = 0; i < warmup_time; i++){
                // BM, BN, BK, TM, TN
                mysgemm_v3<128, 128, 8, 8, 8><<<blocks, threads>>>(N, N, N, alpha, d_A, d_B, beta, d_C_v1);
            }
            hipDeviceSynchronize();

            hipEventRecord(start);
            for (int i = 0; i < repeat_time; i++){
                // BM, BN, BK, TM, TN
                mysgemm_v3<128, 128, 8, 8, 8><<<blocks, threads>>>(N, N, N, alpha, d_A, d_B, beta, d_C_v1);
            }
            hipEventRecord(end);
            hipEventSynchronize(end);

            float v3_time = 0.f;
            hipEventElapsedTime(&v3_time, start, end);

            hipMemcpy(C_v1, d_C_v1, numBytes, hipMemcpyDeviceToHost);
            std::cout<<"v3 time:"<<v3_time<<"ms"<<std::endl;

            
            // 结果比较
            bool isMatch = true;
            for (int i = 0; i < elemCount; i++){
                if (fabsf(C_cublas[i] - C_v1[i]) > 1e-3){
                    isMatch = false;
                    break;
                }
            }
            if (isMatch) std::cout<<"Results Match!"<<std::endl;
            else std::cout<<"Results not Match!"<<std::endl;

            
        }
        catch(...){
            std::cerr << "Out of memory or error during testing size: " << N << std::endl;
        }
    }
    
}