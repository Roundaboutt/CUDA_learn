#include <iostream>
#include "hip/hip_runtime.h"

__global__ void transpose_v1(float* output, float* input, int nx, int ny){
    int id_x = threadIdx.x + blockIdx.x * blockDim.x;
    int id_y = threadIdx.y + blockIdx.y * blockDim.y;

    if (id_x >= nx || id_y >= ny) return;
    output[id_x * ny + id_y] = input[id_y * nx + id_x];
}

void call_v1(float* d_output, float* d_input, int nx, int ny){
    dim3 blockSize(16, 16);
    dim3 gridSize((nx + blockSize.x - 1) / blockSize.x, (ny + blockSize.y - 1) / blockSize.y);
    transpose_v1<<<gridSize, blockSize>>>(d_output, d_input, nx, ny);
}


template <const int SUBX, const int SUBY>
__global__ void transpose_v2(float* output, float* input, int nx, int ny){

    __shared__ float tile[SUBY][SUBX];

    int id_x = threadIdx.x + blockIdx.x * blockDim.x;
    int id_y = threadIdx.y + blockIdx.y * blockDim.y;   
    int thread_in = id_y * nx + id_x;

    // 一个block内的二维坐标转化为一维坐标
    int bidx = threadIdx.y * blockDim.x + threadIdx.x;
    // 把这个一维的坐标重排成转置后的二维坐标
    int idx_row = bidx / blockDim.y;
    int idx_col = bidx % blockDim.y;

    // 转置后矩阵的坐标
    id_x = idx_col + blockIdx.y * blockDim.y;
    id_y = idx_row + blockIdx.x * blockDim.x;

    int thread_out = id_y * ny + id_x;

    if (id_x < nx && id_y < ny){
        tile[threadIdx.y][threadIdx.x] = input[thread_in];
        
        __syncthreads();

        output[thread_out] = tile[idx_col][idx_row];
    }
}

void call_v2(float* d_output, float* d_input, int nx, int ny){
    dim3 blockSize(16, 16);
    dim3 gridSize((nx + blockSize.x - 1) / blockSize.x, (ny + blockSize.y) / blockSize.y);

    transpose_v2<32, 16><<<gridSize, blockSize>>>(d_output, d_input, nx, ny);

}



float* v1_time(int nx, int ny){

    int elemCount = nx * ny;
    int numBytes = elemCount * sizeof(float);

    float* h_input,* h_output;
    h_input = (float*)malloc(numBytes);
    h_output = (float*)malloc(numBytes);

    for (int i = 0; i < elemCount; i++){
        h_input[i] = float(i);
    }


    float* d_input,* d_output;
    hipMalloc((void**)&d_input, numBytes);
    hipMalloc((void**)&d_output, numBytes);

    hipMemcpy(d_input, h_input, numBytes, hipMemcpyHostToDevice);

    int warmup = 10;
    for (int i = 0; i < warmup; i++){
        call_v1(d_output, d_input, nx, ny);        
    }

    int repeat_time = 10;

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start);
    for (int i = 0; i < repeat_time; i++){
        call_v1(d_output, d_input, nx, ny);
    }
    hipEventRecord(end);
    hipEventSynchronize(end);

    float v1_time = 0;
    hipEventElapsedTime(&v1_time, start, end);
    std::cout << "v1 time:" << v1_time << "ms" << std::endl;

    hipDeviceSynchronize();
    hipMemcpy(h_output, d_output, numBytes, hipMemcpyDeviceToHost);

    hipEventDestroy(start);
    hipEventDestroy(end);

    hipFree(d_output);
    hipFree(d_input);
    free(h_input);
    return h_output;  
}

float* v2_time(int nx, int ny){
    
    int elemCount = nx * ny;
    int numBytes = elemCount * sizeof(float);

    float* h_input,* h_output;
    h_input = (float*)malloc(numBytes);
    h_output = (float*)malloc(numBytes);

    for (int i = 0; i < elemCount; i++){
        h_input[i] = float(i);
    }


    float* d_input,* d_output;
    hipMalloc((void**)&d_input, numBytes);
    hipMalloc((void**)&d_output, numBytes);

    hipMemcpy(d_input, h_input, numBytes, hipMemcpyHostToDevice);

    int warmup = 10;
    for (int i = 0; i < warmup; i++){
        call_v2(d_output, d_input, nx, ny);        
    }

    int repeat_time = 10;

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start);
    for (int i = 0; i < repeat_time; i++){
        call_v2(d_output, d_input, nx, ny);
    }
    hipEventRecord(end);
    hipEventSynchronize(end);

    float v2_time = 0;
    hipEventElapsedTime(&v2_time, start, end);
    std::cout << "v2 time:" << v2_time << "ms" << std::endl;

    hipDeviceSynchronize();
    hipMemcpy(h_output, d_output, numBytes, hipMemcpyDeviceToHost);

    hipEventDestroy(start);
    hipEventDestroy(end);

    hipFree(d_output);
    hipFree(d_input);
    free(h_input);
    return h_output;
}

bool isMatch(float* a, float* b, int elemCount){
    for (int i = 0; i < elemCount; i++){
        if (fabsf(a[i] - b[i]) > 1e-5) return false;
    }
    return true;
}

int main(){
    int nx = 4096, ny = 4096;
    int numBytes = nx * ny * sizeof(float);
    float* v1_output = (float*)malloc(numBytes);
    float* v2_output = (float*)malloc(numBytes);
    v1_output = v1_time(nx, ny);
    v2_output = v2_time(nx, ny);

    if (isMatch(v1_output, v2_output, nx * ny)){
        std::cout << "Results Match!" << std::endl;
    }
}

