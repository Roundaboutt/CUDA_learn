#include <iostream>
#include "hip/hip_runtime.h"



float* transpose_cpu(float* output, int nx, int ny){

    float input[nx * ny];
    for (int i = 0; i < nx * ny; i++){
        input[i] = float(i);
    }

    for (int i = 0; i < ny; i++){
        for (int j = 0; j < nx; j++){
            // output[j][i] = input[i][j];
            output[j * ny + i] = input[i * nx + j];
        }
    }
    return output;
}


__global__ void transpose_v1(float* output, float* input, int nx, int ny){
    int id_x = threadIdx.x + blockIdx.x * blockDim.x;
    int id_y = threadIdx.y + blockIdx.y * blockDim.y;

    if (id_x >= nx || id_y >= ny) return;
    output[id_x * ny + id_y] = input[id_y * nx + id_x];
}


template <const int BDIM, const int BDIN>
__global__ void transpose_v2(float* output, float* input, int nx, int ny) {

    __shared__ float tile[BDIN][BDIM];

    int id_x = threadIdx.x + blockIdx.x * blockDim.x;
    int id_y = threadIdx.y + blockIdx.y * blockDim.y;
    int thread_in = id_y * nx + id_x;

    int bidx = threadIdx.y * blockDim.x + threadIdx.x;
    int idx_row = bidx / blockDim.y;
    int idx_col = bidx % blockDim.y;

    // 转置后 block 对换位置
    int trans_x = blockIdx.y * blockDim.y + idx_col;
    int trans_y = blockIdx.x * blockDim.x + idx_row;
    int thread_out = trans_y * ny + trans_x;

    if (id_x < nx && id_y < ny) {
        tile[threadIdx.y][threadIdx.x] = input[thread_in];
    }

    __syncthreads();

    if (trans_x < ny && trans_y < nx) {
        output[thread_out] = tile[idx_col][idx_row];
    }
}


void call_v1(float* d_output, float* d_input, int nx, int ny){
    dim3 blockSize(32, 16);
    dim3 gridSize((nx + blockSize.x - 1) / blockSize.x, (ny + blockSize.y - 1) / blockSize.y);
    transpose_v1<<<gridSize, blockSize>>>(d_output, d_input, nx, ny);
}

void call_v2(float* d_output, float* d_input, int nx, int ny){
    dim3 blockSize(32, 16);
    dim3 gridSize((nx + blockSize.x - 1) / blockSize.x, (ny + blockSize.y) / blockSize.y);

    transpose_v2<32, 16><<<gridSize, blockSize>>>(d_output, d_input, nx, ny);

}



float* v1_time(int nx, int ny){

    int elemCount = nx * ny;
    int numBytes = elemCount * sizeof(float);

    float* h_input,* h_output;
    h_input = (float*)malloc(numBytes);
    h_output = (float*)malloc(numBytes);

    for (int i = 0; i < elemCount; i++){
        h_input[i] = float(i);
    }


    float* d_input,* d_output;
    hipMalloc((void**)&d_input, numBytes);
    hipMalloc((void**)&d_output, numBytes);

    hipMemcpy(d_input, h_input, numBytes, hipMemcpyHostToDevice);

    int warmup = 10;
    for (int i = 0; i < warmup; i++){
        call_v1(d_output, d_input, nx, ny);        
    }

    int repeat_time = 20;

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start);
    for (int i = 0; i < repeat_time; i++){
        call_v1(d_output, d_input, nx, ny);
    }
    hipEventRecord(end);
    hipEventSynchronize(end);

    float v1_time = 0;
    hipEventElapsedTime(&v1_time, start, end);
    std::cout << "v1 time:" << v1_time << "ms" << std::endl;

    hipDeviceSynchronize();
    hipMemcpy(h_output, d_output, numBytes, hipMemcpyDeviceToHost);

    hipEventDestroy(start);
    hipEventDestroy(end);

    hipFree(d_output);
    hipFree(d_input);
    free(h_input);
    return h_output;  
}

float* v2_time(int nx, int ny){
    
    int elemCount = nx * ny;
    int numBytes = elemCount * sizeof(float);

    float* h_input,* h_output;
    h_input = (float*)malloc(numBytes);
    h_output = (float*)malloc(numBytes);

    for (int i = 0; i < elemCount; i++){
        h_input[i] = float(i);
    }


    float* d_input,* d_output;
    hipMalloc((void**)&d_input, numBytes);
    hipMalloc((void**)&d_output, numBytes);

    hipMemcpy(d_input, h_input, numBytes, hipMemcpyHostToDevice);

    int warmup = 10;
    for (int i = 0; i < warmup; i++){
        call_v2(d_output, d_input, nx, ny);        
    }

    int repeat_time = 20;

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start);
    for (int i = 0; i < repeat_time; i++){
        call_v2(d_output, d_input, nx, ny);
    }
    hipEventRecord(end);
    hipEventSynchronize(end);

    float v2_time = 0;
    hipEventElapsedTime(&v2_time, start, end);
    std::cout << "v2 time:" << v2_time << "ms" << std::endl;

    hipDeviceSynchronize();
    hipMemcpy(h_output, d_output, numBytes, hipMemcpyDeviceToHost);

    hipEventDestroy(start);
    hipEventDestroy(end);

    hipFree(d_output);
    hipFree(d_input);
    free(h_input);
    return h_output;
}

bool isMatch(float* a, float* b, int elemCount){
    for (int i = 0; i < elemCount; i++){
        if (fabsf(a[i] - b[i]) > 1e-5) return false;
    }
    return true;
}

int main(){
    int nx = 1024, ny = 512;
    int numBytes = nx * ny * sizeof(float);

    float* cpu_output = (float*)malloc(numBytes);    
    float* v1_output = (float*)malloc(numBytes);
    float* v2_output = (float*)malloc(numBytes);

    cpu_output = transpose_cpu(cpu_output, nx, ny);
    v1_output = v1_time(nx, ny);
    v2_output = v2_time(nx, ny);

    if (isMatch(cpu_output, v2_output, nx * ny)){
        std::cout << "Results Match!" << std::endl;
    }
    else{
        std::cout << "Results not Match!" << std::endl;
    }
}

