#include<iostream>
#include"hip/hip_runtime.h"
#include<vector>
#include <hipblas.h>



template <const int BLOCKSIZE>

__global__ void mysgemm_v2(int M, int N, int K, float alpha, float* A, float* B, float beta, float* C){
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // 为每个线程块分配一个矩阵中的小分块进行处理
    const int BM = BLOCKSIZE;
    const int BN = BLOCKSIZE;
    const int BK = BLOCKSIZE;

    __shared__ float As[BM * BK];
    __shared__ float Bs[BK * BN];
    
    int tx = threadIdx.x % BN;
    int ty = threadIdx.x / BN;

    A = &A[by * BM * BK];
    B = &B[bx * BN];
    C = &C[by * BM * N + bx * BN];

    float temp = 0.f;
    for (int k = 0; k < K; k += BK){
        As[ty * BK + tx] = A[ty * K + tx];
        Bs[ty * BN + tx] = B[ty * N + tx];

        __syncthreads();

        A += BK;
        B += BK * N;

        for (int i = 0; i < BK; i++){
            temp += As[ty * BK + i] * Bs[i * BN + tx];
        }

        __syncthreads();

        C[ty * N + tx] = alpha * temp + beta * C[ty * N + tx];
    }
}

int main(){
    std::vector<int> sizes = {128, 256, 512, 1024, 2048, 4096, 8192};
    for (int N:sizes){
        size_t elemCount = N * N;
        std::cout<<"------------------------Testing size: "<< N <<"------------------------"<< std::endl;

        size_t numBytes = elemCount * sizeof(float);
        float* A = (float*)malloc(numBytes);
        float* B = (float*)malloc(numBytes);
        float* C_cublas = (float*)malloc(numBytes);
        float* C_v1 = (float*)malloc(numBytes);

        float* d_A,* d_B, * d_C_v1;
        hipMalloc(&d_A, numBytes);
        hipMalloc(&d_B, numBytes);
        hipMalloc(&d_C_v1, numBytes);

        try{
            for (int i = 0; i < elemCount; i++){
                A[i] = 1.0f;
                B[i] = 2.0f;
            }

            hipMemcpy(d_A, A, numBytes, hipMemcpyHostToDevice);
            hipMemcpy(d_B, B, numBytes, hipMemcpyHostToDevice);
            
            hipblasHandle_t handle;  //定义句柄, 用于记录状态
            hipblasCreate(&handle);  // 创建cuBLAS上下文

            float alpha = 1.f;
            float beta = 0.f;
            

            /*------------------------cublas计算/*------------------------*/
            hipEvent_t start, end;
            hipEventCreate(&start);
            hipEventCreate(&end);

            int warmup_time = 10;
            for (int i = 0; i < warmup_time; i++){
                hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_B, N, d_A, N, &beta, d_C_v1, N);
            }

            hipDeviceSynchronize();

            int repeat_time = 5;
            hipEventRecord(start);
            for (int i = 0; i < repeat_time; i++){
                hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_B, N, d_A, N, &beta, d_C_v1, N);
            }
            hipEventRecord(end);
            hipEventSynchronize(end);

            float cublas_time = 0;
            hipEventElapsedTime(&cublas_time, start, end);
            
            hipMemcpy(C_cublas, d_C_v1, numBytes, hipMemcpyDeviceToHost);
            std::cout<<"cublas time:"<<cublas_time<<"ms"<<std::endl;

            /*------------------------v1计算------------------------*/
            dim3 threads(1024); // 1024拆成了32x32
            dim3 blocks((N + 32 - 1) / 32, (N + 32 - 1) / 32);
            
            for (int i = 0; i < warmup_time; i++){
                mysgemm_v2<32><<<blocks, threads>>>(N, N, N, alpha, d_A, d_B, beta, d_C_v1);
            }
            hipDeviceSynchronize();

            hipEventRecord(start);
            for (int i = 0; i < repeat_time; i++){
                mysgemm_v2<32><<<blocks, threads>>>(N, N, N, alpha, d_A, d_B, beta, d_C_v1);
            }
            hipEventRecord(end);
            hipEventSynchronize(end);

            float v1_time = 0.f;
            hipEventElapsedTime(&v1_time, start, end);

            hipMemcpy(C_v1, d_C_v1, numBytes, hipMemcpyDeviceToHost);
            std::cout<<"v2 time:"<<v1_time<<"ms"<<std::endl;

            
            // 结果比较
            bool isMatch = true;
            for (int i = 0; i < elemCount; i++){
                if (fabsf(C_cublas[i] - C_v1[i]) > 1e-3){
                    isMatch = false;
                    break;
                }
            }
            if (isMatch) std::cout<<"Results Match!"<<std::endl;
            else std::cout<<"Results not Match!"<<std::endl;

            
        }
        catch(...){
            std::cerr << "Out of memory or error during testing size: " << N << std::endl;
        }
    }
    
}