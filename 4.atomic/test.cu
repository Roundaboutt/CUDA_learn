#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void non_atomic_add(int* data) {
    int temp = *data;
    temp = temp + 1;
    *data = temp;
}

__global__ void atomic_add(int* data){
    atomicAdd(data, 1);
}

int main() {
    int *d_data1,* d_data2;
    int h_data1 = 0, h_data2 = 0;

    hipMalloc(&d_data1, sizeof(int));
    hipMalloc(&d_data2, sizeof(int));
    hipMemcpy(d_data1, &h_data1, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_data2, &h_data1, sizeof(int), hipMemcpyHostToDevice);

    non_atomic_add<<<1024, 256>>>(d_data1);
    atomic_add<<<1024, 256>>>(d_data2);

    hipMemcpy(&h_data1, d_data1, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&h_data2, d_data2, sizeof(int), hipMemcpyDeviceToHost);

    printf("Final value: %d (expected %d if no race condition)\n", h_data1, 1024 * 256);
    printf("Final value: %d (expected %d if race condition)\n", h_data2, 1024 * 256);

    hipFree(d_data1);
    hipFree(d_data2);
    return 0;
}
