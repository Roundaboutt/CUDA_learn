#include <iostream>
#include "hip/hip_runtime.h"



float* transpose_cpu(float* output, int nx, int ny){

    float* input = (float*)malloc(sizeof(float) * nx * ny);
    for (int i = 0; i < nx * ny; i++){
        input[i] = float(i);
    }

    for (int i = 0; i < ny; i++){
        for (int j = 0; j < nx; j++){
            // output[j][i] = input[i][j];
            output[j * ny + i] = input[i * nx + j];
        }
    }
    return output;
}


// 共享显存优化
template <const int BLKDIM_X, const int BLKDIM_Y, const int PAD>
__global__ void transpose_v2(float* output, float* input, int M, int N) {

    __shared__ float tile[BLKDIM_Y][BLKDIM_X + PAD];

    // 线程在整个矩阵中的二维坐标
    int id_x = threadIdx.x + blockIdx.x * blockDim.x;
    int id_y = threadIdx.y + blockIdx.y * blockDim.y;
    // 线程在整个矩阵中的索引(input)
    int thread_in = id_y * M + id_x;

    // 线程在一个block(tile)内的索引
    int bidx = threadIdx.y * blockDim.x + threadIdx.x;
    // 线程在一个block(tile)内的二维坐标(转置后)
    int idx_row = bidx / blockDim.y;
    int idx_col = bidx % blockDim.y;

    // 转置后在整个矩阵中的坐标
    int trans_x = blockIdx.y * blockDim.y + idx_col;
    int trans_y = blockIdx.x * blockDim.x + idx_row;
    //转化为output中的一维索引
    int thread_out = trans_y * N + trans_x;

    // input -> tile
    if (id_x < M && id_y < N) {
        tile[threadIdx.y][threadIdx.x] = input[thread_in];
    }

    __syncthreads();

    // tile -> output
    if (trans_x < N && trans_y < M) {
        output[thread_out] = tile[idx_col][idx_row];
    }
}

template <const int BLKDIM_X, const int BLKDIM_Y, const int PAD>
void call_v2(float* d_output, float* d_input, int M, int N){
    dim3 blockSize(BLKDIM_X, BLKDIM_Y);
    dim3 gridSize((M + blockSize.x - 1) / blockSize.x, (N + blockSize.y) / blockSize.y);

    transpose_v2<BLKDIM_X, BLKDIM_Y, PAD><<<gridSize, blockSize>>>(d_output, d_input, M, N);

}

template <const int BLKDIM_X, const int BLKDIM_Y, const int PAD>
float* v2_time(int M, int N, int warmup, int repeat_time){
    
    int elemCount = M * N;
    int numBytes = elemCount * sizeof(float);

    float* h_input,* h_output;
    h_input = (float*)malloc(numBytes);
    h_output = (float*)malloc(numBytes);

    for (int i = 0; i < elemCount; i++){
        h_input[i] = float(i);
    }


    float* d_input,* d_output;
    hipMalloc((void**)&d_input, numBytes);
    hipMalloc((void**)&d_output, numBytes);

    hipMemcpy(d_input, h_input, numBytes, hipMemcpyHostToDevice);

    for (int i = 0; i < warmup; i++){
        call_v2<BLKDIM_X, BLKDIM_Y, PAD>(d_output, d_input, M, N);        
    }


    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start);
    for (int i = 0; i < repeat_time; i++){
        call_v2<BLKDIM_X, BLKDIM_Y, PAD>(d_output, d_input, M, N);
    }
    hipEventRecord(end);
    hipEventSynchronize(end);

    float v2_time = 0;
    hipEventElapsedTime(&v2_time, start, end);
    std::cout << "v2 time:" << v2_time << "ms" << std::endl;

    hipDeviceSynchronize();
    hipMemcpy(h_output, d_output, numBytes, hipMemcpyDeviceToHost);

    hipEventDestroy(start);
    hipEventDestroy(end);

    hipFree(d_output);
    hipFree(d_input);
    free(h_input);
    return h_output;
}

bool isMatch(float* a, float* b, int elemCount){
    for (int i = 0; i < elemCount; i++){
        if (fabsf(a[i] - b[i]) > 1e-5) return false;
    }
    return true;
}

int main(){
    int M = 4096, N = 4096;
    int numBytes = M * N * sizeof(float);


    float* cpu_output = (float*)malloc(numBytes);    
    float* v1_output = (float*)malloc(numBytes);
    float* v2_output = (float*)malloc(numBytes);

    cpu_output = transpose_cpu(cpu_output, M, N);
    v2_output = v2_time<32, 32, 0>(M, N, 10, 10);
    v2_output = v2_time<32, 32, 1>(M, N, 10, 10);
    v2_output = v2_time<32, 32, 2>(M, N, 10, 10);

    if (isMatch(cpu_output, v2_output, M * N)){
        std::cout << "Results Match!" << std::endl;
    }
    else{
        std::cout << "Results not Match!" << std::endl;
    }
}

