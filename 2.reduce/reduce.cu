#include <stdio.h>
#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <chrono>


const int BLOCK_SIZE = 1024;
const int N = 1024 * 1024;

__global__ void reduce_v0(float* input, float* output){
    __shared__ float shared[BLOCK_SIZE];

    unsigned int tid = threadIdx.x;
    unsigned int bid = blockIdx.x;
    unsigned int id = tid + blockDim.x * bid;

    if (id < N){
        shared[tid] = input[id];    // 把当前线程的元素复制到共享内存        
    }
    else{
        shared[tid] = 0.f;
    }


    for (unsigned int s = 1; s < blockDim.x; s *= 2){
        __syncthreads();
        if (tid % (2 * s) == 0){
            shared[tid] += shared[tid + s]; 
        }
        
    }
    
    __syncthreads();
    // 该线程块的局部规约结果
    if (tid == 0) output[bid] = shared[0]; 
}


float reduce_cpu(const std::vector<float> &data){
    float sum = 0.f;
    for (float val : data){
        sum += val;
    }
    return sum;
}


int main(){

    int num_blocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    std::vector<float> h_data(N);

    for (int i = 0; i < N; i++){
        h_data[i] = 1.f;
    }

    auto cpu_start = std::chrono::high_resolution_clock::now();
    float cpu_res = reduce_cpu(h_data);
    auto cpu_end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> cpu_duration = cpu_end - cpu_start;

    std::cout<<"cpu result:"<<cpu_res<<std::endl;
    std::cout<<"cpu time:"<<cpu_duration.count()<<"ms"<<std::endl;



    float* d_input,* d_output;
    float* d_final_output;
    float gpu_res;

    hipMalloc((void**)&d_input, N * sizeof(float));    // 输入的数组
    hipMalloc((void**)&d_output, num_blocks * sizeof(float));  // 第一步归约的结果,计算每个block的总和
    hipMalloc((void**)&d_final_output, 1 * sizeof(float));     //最后归约的结果,所有block的和
    hipMemcpy(d_input, h_data.data(), N * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    reduce_v0<<<num_blocks, BLOCK_SIZE>>>(d_input, d_output);
    reduce_v0<<<1, num_blocks>>>(d_output, d_final_output);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    



    float gpu_time = 0;
    hipEventElapsedTime(&gpu_time, start, stop);
    hipMemcpy(&gpu_res, d_final_output, sizeof(float), hipMemcpyDeviceToHost);  

    std::cout<<"gpu res:"<<gpu_res<<std::endl;
    std::cout<<"gpu time:"<<gpu_time<<"ms"<<std::endl;
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_output);
    hipFree(d_final_output);
    hipFree(d_input);
}