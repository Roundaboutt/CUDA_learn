#include<iostream>
#include<chrono>
#include"hip/hip_runtime.h"

#define N 1024 * 1024
#define BLOCKSIZE 1024

__global__ void reduce_v2(float* input, float* output){
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int id = tid + bid * blockDim.x * 2;

    __shared__ float shared[BLOCKSIZE];
    shared[tid] = input[id] + input[id + blockDim.x];

    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1){
        if (tid < s){
            shared[tid] += shared[tid + s];            
        }
        __syncthreads();
    }

    if (tid == 0) output[bid] = shared[0];
}

float reduce_cpu(float* input){
    float sum = 0.0f;
    for (int i = 0;i < N; i++){
        sum += input[i];
    }
    return sum;
}

int main(){
    const int num_blocks = ((N + BLOCKSIZE - 1) / BLOCKSIZE) / 2;

    const int elemCount = N;
    const int numBytes = N * sizeof(float);

    float* h_input = (float*)malloc(numBytes);

    for (int i = 0;i < elemCount; i++){
        h_input[i] = 1.0f;
    }
    
    auto cpu_start = std::chrono::high_resolution_clock::now();
    float cpu_res = reduce_cpu(h_input);
    auto cpu_end = std::chrono::high_resolution_clock::now();    
    std::chrono::duration<double, std::milli> cpu_time = cpu_end - cpu_start;

    std::cout<<"cpu result:"<<cpu_res<<std::endl;
    std::cout<<"cpu time:"<<cpu_time.count()<<"ms"<<std::endl;

    float* d_input,* d_output,* d_final_res;
    float gpu_res;
    hipMalloc((void**)&d_input, numBytes);
    hipMalloc((void**)&d_output, numBytes);
    hipMalloc((void**)&d_final_res, sizeof(float));

    hipMemcpy(d_input, h_input, numBytes, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    reduce_v2<<<num_blocks, BLOCKSIZE>>>(d_input, d_output);
    reduce_v2<<<1, num_blocks>>>(d_output, d_final_res);
    hipEventRecord(stop);

    hipEventSynchronize(stop);

    float gpu_time = 0;
    hipEventElapsedTime(&gpu_time, start, stop);

    hipMemcpy(&gpu_res, d_final_res, sizeof(float), hipMemcpyDeviceToHost);

    std::cout<<"gpu result:"<<gpu_res<<std::endl;
    std::cout<<"gpu time:"<<gpu_time<<"ms"<<std::endl;

    if (abs(cpu_res - gpu_res) < 1e-5) {
        std::cout << "Result verified successfully!" << std::endl;
    } else {
        std::cout << "Result verification failed!" << std::endl;
    }

    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_final_res);

    free(h_input);
    
}